
#include <hip/hip_runtime.h>
////////////////////////////////////////////////////////////////////////////////
// vectorScalar kernel
////////////////////////////////////////////////////////////////////////////////

__global__ void vectorScalarProduct(float *vector_d, float *wector_d, float *scalar_d, int n)
{
    extern __shared__ float shared[];
    
    // global thread ID in grid
    int tidg = blockIdx.x * blockDim.x + threadIdx.x;

    
    if(tidg < n) {
        shared[tidg] = vector_d[tidg] * wector_d[tidg];   
        scalar_d[tidg] = shared[tidg];
    }


}
