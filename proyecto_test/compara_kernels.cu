/* -------------------------------------------------------------------------- */
/* Project: I Curso de Computación Científica en Clusters                     */
/* Author:  Juan Fernández Peinador                                           */
/* Date:    Marzo de 2010                                                     */
/* Actualizado en Febrero 2021 para cuda 8.0: hipDeviceReset()		      */
/* -------------------------------------------------------------------------- */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// ayuda con los ejemplos
// These are CUDA Helper functions for initialization and error checking
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <timer.h>

////////////////////////////////////////////////////////////////////////////////

// includes, kernels
#include "compara_kernels_kernel.cu"

#define TEST
// #define DEBUG

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

void print_matrix(float *m, int t1, int t2)
{
    for(int i = 0; i < t1; i++)
    {
        for(int j = 0; j < t2; j++)
            printf("%f ", m[i * t2 + j]);
        printf("\n");
    }
}


float * multiply(float *A, float *B,  float *res, int m, int n, int w)
{
    float *C =(float*) malloc(m * n * sizeof(float));

    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            C[i * n + j] = 0.0f;
            for (int k = 0; k < w; k++)
            {
                C[i * n + j] += A[i * w + k] * B[k * n + j];
            }

            // assert(C[i * n + j] == res[i * n + j]);
            assert(C[i * n + j] - res[i * n + j] <= 1e-3);
            
        }
    }
    return C;
}

void test(float *A, float *B,  float *res, int m, int n, int w)
{

    

    float *host = multiply(A, B, res, m, n, w);


#ifdef DEBUG
    printf("A: \n");
    print_matrix(A, m, w);

    printf("B: \n");
    print_matrix(B, w, n);

    printf("CUDA: \n");
    print_matrix(res, m, n);

    printf("HOST SECUENTIAL\n");
    print_matrix(host, m, n);
#endif
}

int main(int argc, char **argv)
{
    float *h_A, *h_B, *h_C; // host data
    float *d_A, *d_B, *d_C; // device data
    size_t size;
    size_t nBytes;

    // default values
    int dim_mat = 1;   // n
    int dim_block = 1; // w

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // events
    float processing_time;
    hipEvent_t start_event, stop_event;

    // process command line arguments
    dim_mat = getCmdLineArgumentInt(argc, (const char **)argv, (const char *)"N") ?: dim_mat;
    dim_block = getCmdLineArgumentInt(argc, (const char **)argv, (const char *)"W") ?: dim_block;

    assert(dim_mat % dim_block == 0);

    size = dim_mat * dim_mat;

    nBytes = size * sizeof(float);
    int t = dim_mat / dim_block;

    // setup execution parameters
    dim3 grid(t, t);
    dim3 block(dim_block, dim_block);

    // allocate host memory
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    h_C = (float *)calloc(size, sizeof(float));

    for (int i = 0; i < size_AB; i++)
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // allocate device memory
    checkCudaErrors(hipMalloc((void **)&d_A, nBytes));
    checkCudaErrors(hipMalloc((void **)&d_B, nBytes));
    checkCudaErrors(hipMalloc((void **)&d_C, nBytes));

    // copy data from host memory to device memory
    checkCudaErrors(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(d_C, 0, nBytes));

    // execute the kernel
    printf("Running configuration: grid of %dx%d blocks of %dx%d threads (%d threads) - KERNEL: %d\n",
           grid.x, grid.y, block.x, block.y, grid.x * grid.y * block.x * block.y, kernel);

    // create events
    checkCudaErrors(hipEventCreate(&start_event, 0));
    checkCudaErrors(hipEventCreate(&stop_event, 0));

    // using events
    checkCudaErrors(hipEventRecord(start_event, 0));


    sharedABMultiply<<<grid, block, 2 * dim_block * dim_block * sizeof(float)>>>(d_A, d_B, d_C, dim_mat, dim_block);

    // wait for thread completion
    hipDeviceSynchronize();

    // ///*using event*/
    checkCudaErrors(hipEventRecord(stop_event, 0));
    hipEventSynchronize(stop_event); // block until the event is actually recorded
    checkCudaErrors(hipEventElapsedTime(&processing_time, start_event, stop_event));
    printf("Processing time: %f (ms)\n", processing_time);

    checkCudaErrors(hipMemcpy(h_C, d_C, nBytes, hipMemcpyDeviceToHost));

#ifdef TEST
    // check result
    test(h_A, h_B, h_C, dim_mat, dim_mat, dim_block);
#endif
    // free memory
    free(h_A);
    free(h_B);
    free(h_C);
    checkCudaErrors(hipFree((void *)d_A));
    checkCudaErrors(hipFree((void *)d_B));
    checkCudaErrors(hipFree((void *)d_C));

    printf("\nTest PASSED\n");

    //    hipDeviceReset();

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
